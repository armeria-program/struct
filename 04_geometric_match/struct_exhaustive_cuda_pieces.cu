#include "hip/hip_runtime.h"
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/system_error.h>



#ifdef	__cplusplus
extern "C" {
#endif

#include <hip/hip_runtime.h>   
#include "struct.h"

#define TILE_WIDTH 16
#define PITCH 64
#define MEM_SIZE_MAX 134217728L
#define CLOCK_PRECISION  1E9
    
struct greater_rmsd{
    
    __host__ __device__
    bool operator()(Triple x, Triple y) 
    {
        return x.rmsd < y.rmsd;
    }
            
};
    
    
__device__ int  sum_to_zero_gpu (float *a, float *b ) {
    int i;
    float sum,aux;

    sum = 0;
    for (i=0; i<3; i++ ) {
	aux = a[i] + b[i];
	sum += aux*aux;
    }
    return (sum < 0.001);
}


__device__ int  normalized_cross_gpu (float *x, float *y, float * v, float *norm_ptr) {

    /* v is the output */
    float norm = 0;
    float vec[3];
    int i;

    if (sum_to_zero_gpu (x, y) ) return 1;
    
    vec[0] = x[1]*y[2] -  x[2]*y[1];
    norm += vec[0]*vec[0];
    vec[1] = x[2]*y[0] -  x[0]*y[2]; 
    norm += vec[1]*vec[1];
    vec[2] = x[0]*y[1] -  x[1]*y[0];
    norm += vec[2]*vec[2];
    norm = sqrt (norm);

    for (i=0; i<3; i++ ) {
	v[i] = vec[i] /norm;
    }

    if ( norm_ptr) *norm_ptr = norm;
    

    return 0;
    
}

__device__ int unnorm_dot_gpu (float *x, float *y, float * dot) {


    float cosine = 0;
    int i;
    
    for (i=0; i<3; i++ ) {
	cosine += x[i]*y[i];
    }

    if (cosine > 1.0 ) 
	cosine = 1.0; /* this should be numerical */
   
    *dot = cosine;
   
    return 0;
    
}


__device__ int distance_of_nearest_approach_gpu(float *x_d, float *x_cm_d, float *y_d, float *y_cm_d, int *set_of_directions_x,  int *set_of_directions_y,
        int set_size, float * rmsd_ptr) {

    float cm_vector[3], distance_x, distance_y;
    float aux, rmsd;
    int i, ray_a, ray_b, a, b, prev_next, norm;
    float cross[3];

    //if (set_size <= 1) return 1;
       
    float * row_data_a, * row_data_b;

    
    rmsd = 0.0;
    norm = 0;
    /* the rmsd for the remaining vectors is ... */
    for (a = 0; a < set_size; a++) {

        for (prev_next = -1; prev_next <= +1; prev_next += 2) {
            b = (set_size + a + prev_next) % set_size;

            ray_a = set_of_directions_x[a];
            ray_b = set_of_directions_x[b];
            
            row_data_a = (float*)(((char*)x_cm_d) + (ray_a * PITCH));
            row_data_b = (float*)(((char*)x_cm_d) + (ray_b * PITCH));
    
            
            /* distance of nearest approach of ray b
               to the cm of a, in the set of directions x */
            
            for (i = 0; i < 3; ++i) {
                cm_vector[i] = row_data_b[i] - row_data_a[i];
            }
            
            
            row_data_a = (float*)(((char*)x_d) + (ray_a * PITCH));
            normalized_cross_gpu(cm_vector, row_data_a, cross, &distance_x);

            ray_a = set_of_directions_y[a];
            ray_b = set_of_directions_y[b];
            /* distance of nearest approach of ray b
               to the cm of a, in the set of directions y */
            
            row_data_a = (float*)(((char*)y_cm_d) + (ray_a * PITCH));
            row_data_b = (float*)(((char*)y_cm_d) + (ray_b * PITCH));
    
            for (i = 0; i < 3; ++i) {
                cm_vector[i] = row_data_b[i] - row_data_a[i];
            }
            
            row_data_a = (float*)(((char*)y_d) + (ray_a * PITCH));
            normalized_cross_gpu(cm_vector, row_data_a, cross, &distance_y);

            aux = distance_x - distance_y;
            rmsd += aux*aux;
            norm++;
        }

    }

    rmsd /= norm;
    rmsd = sqrt(rmsd);

    *rmsd_ptr = rmsd;

    return 0;
}

__device__ int  same_hand_triple_gpu(float *x_d, float *x_cm_d, float *y_d, float *y_cm_d, int *set_of_directions_x,
        int *set_of_directions_y, int set_size) {

    float cm_vector[3], avg_cm[3], cross[3], dx, dy;
    int i, ray_a, ray_b, ray_c;

    // if (set_size != 3) return 0;

    
    /*****************************/
    /*****************************/
    ray_a = set_of_directions_x[0];
    ray_b = set_of_directions_x[1];
    ray_c = set_of_directions_x[2];
    /* I better not use the cross prod here: a small diff
       int he angle makeing them pointing toward each other or
       away from each other changes the direction of cross prod;
       rather, use one vector, and distance between the cm's as the other */
    float * row_data_a, * row_data_b, * row_data_c;

    row_data_a = (float*)(((char*)x_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)x_d) + (ray_b * PITCH));
   
    
    normalized_cross_gpu(row_data_a, row_data_b, cross, NULL);
    
    row_data_a = (float*)(((char*)x_cm_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)x_cm_d) + (ray_b * PITCH));
    row_data_c = (float*)(((char*)x_cm_d) + (ray_c * PITCH)); 
    
    /* note I am making another cm vector here */
    for (i = 0; i < 3; i++) {
        avg_cm[i] = (row_data_b[i] - row_data_a[i])/2;
        cm_vector[i] = row_data_c[i] - avg_cm[i];
        //avg_cm[i] = (x_cm_d[ray_b][i] + x_cm_d[ray_a][i]) / 2;
        //cm_vector[i] = x_cm_d[ray_c][i] - avg_cm[i];
    }
    unnorm_dot_gpu(cm_vector, cross, &dx);

    /*****************************/
    /*****************************/
    ray_a = set_of_directions_y[0];
    ray_b = set_of_directions_y[1];
    ray_c = set_of_directions_y[2];
    
    row_data_a = (float*)(((char*)y_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)y_d) + (ray_b * PITCH));
   
    normalized_cross_gpu(row_data_a, row_data_b, cross, NULL);
    
    row_data_a = (float*)(((char*)y_cm_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)y_cm_d) + (ray_b * PITCH));
    row_data_c = (float*)(((char*)y_cm_d) + (ray_c * PITCH));    
    
    /* note I am making another cm vector here */
    for (i = 0; i < 3; i++) {
        avg_cm[i] = (row_data_b[i] - row_data_a[i])/2;
        cm_vector[i] = row_data_c[i]- avg_cm[i];
        
        //avg_cm[i] = (y_cm_d[ray_b][i] + y_cm_d[ray_a][i]) / 2;
        //cm_vector[i] = y_cm_d[ray_c][i] - avg_cm[i];
    }
    /*note: unnorm_dot thinks it is getting unit vectors,
      and evrything that is >1 will be "rounded" to 1
      (similarly for -1) - it doesn't do the normalization itself*/
    
    unnorm_dot_gpu(cm_vector, cross, &dy);

    if (dx * dy < 0) return 0;


    return 1; /* this isn't err value - the handedness is the same */
}


__global__ void find_triplets_gpu(float *x_d, float *x_cm_d, float *y_d, float *y_cm_d, int NX, int NY, int *x_triple_array_d, int *y_triple_array_d, 
        int cnt_x, int cnt_y, Triple *triple_array_d, float * rmsd_array_d){
    
    int row = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int col = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int i;

    
    int *row_data;
    float rmsd = -5; 
    
    int triple_x[3], triple_y[3];
    // double q_init[4] = {0.0}; // no change

    
    if (row < cnt_x && col < cnt_y) {
        triple_array_d[row*cnt_y + col].rmsd = BAD_RMSD + 1;
        rmsd_array_d[row*cnt_y + col] = BAD_RMSD + 1;
        row_data = (int*)(((char*)x_triple_array_d) + (row * PITCH));
        
        for (i = 0; i < 3; ++i) {
            triple_x[i] =  row_data[i];
        }
        
        row_data = (int*)(((char*)y_triple_array_d) + (col * PITCH));
        
        for (i = 0; i < 3; ++i) {
            triple_y[i] =  row_data[i];
        }  
        
        for (i = 0; i <3; ++i) {
            triple_array_d[row*cnt_y + col].triple_x[i] = triple_x[i];
            triple_array_d[row*cnt_y + col].triple_y[i] = triple_y[i];
        }

        //if (!same_hand_triple_gpu(x_d, x_cm_d, y_d, y_cm_d, triple_x,triple_y, 3) ) return;

        distance_of_nearest_approach_gpu(x_d, x_cm_d, y_d, y_cm_d, triple_x, triple_y, 3, &rmsd);
        if (rmsd > BAD_RMSD) {
            return; 
        }
                    // insert values to array of structs
            
        triple_array_d[row*cnt_y + col].rmsd = rmsd;
        rmsd_array_d[row*cnt_y + col] = rmsd;
        
    }
    

/*
   if (!same_hand_triple_gpu(x_d, x_cm_d, y_d, y_cm_d, triple_x,triple_y, 3) ) return;
*/
          // if (!same_hand_triple(X_rep, x_triple, Y_rep, y_triple, 3)) continue;

/*
    distance_of_nearest_approach_gpu(x_d, x_cm_d, y_d, y_cm_d, triple_x, triple_y, 3, &rmsd);
    if (rmsd > BAD_RMSD) {
        return; 
    }
*/



            // insert values to array of structs
            
//    triple_array_d[row*cnt_y + col].rmsd = rmsd;
            //printf("%d\n", myid);

    
}

extern int insert_triple_to_heap_gpu(Representation* X_rep, Representation* Y_rep,
				     int ** x_triple_array, int ** y_triple_array, int x_triple_cnt,
				     int y_triple_cnt, PriorityQueue * heap) {
    

    double **x_db = X_rep->full;
    double **x_cm_db = X_rep->cm;
    double **y_db = Y_rep->full;
    double **y_cm_db = Y_rep->cm;
    int NX = X_rep->N_full;
    int NY = Y_rep->N_full;
    
    float **x = fmatrix(NX, 3);
    float **x_cm = fmatrix(NX, 3);;
    float **y = fmatrix(NY, 3);;
    float **y_cm = fmatrix(NY, 3);;
    
    int i,j;
    
    for(i = 0; i < NX; ++i) {
        for(j = 0; j < 3; ++j) {
            x[i][j] = x_db[i][j];
            x_cm[i][j] = x_cm_db[i][j];
        }
    }
    
    for(i = 0; i < NY; ++i) {
        for(j = 0; j < 3; ++j) {
            y[i][j] = y_db[i][j];
            y_cm[i][j] = y_cm_db[i][j];
        }
    }
    
    float *x_d;
    float *x_cm_d;
    float *y_d;
    float *y_cm_d;
    
    int *x_triple_array_d;
    int *y_triple_array_d;
    
    
    int cnt_x = x_triple_cnt;
    int cnt_y = y_triple_cnt;
    
    struct timespec requestStart, requestEnd;
    clock_gettime(CLOCK_REALTIME, &requestStart);
        
    
    Triple * triple_array = (Triple *) malloc(TOP_RMSD * sizeof(Triple));
    
    
    if (hipSuccess != hipMalloc(&x_d, NX * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(x_d,PITCH,x[0],sizeof(float)*3,sizeof(float)*3,NX,hipMemcpyHostToDevice);   
    
    if (hipSuccess != hipMalloc(&x_cm_d, NX * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(x_cm_d,PITCH,x_cm[0],sizeof(float)*3,sizeof(float)*3,NX,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&y_d, NY * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(y_d,PITCH,y[0],sizeof(float)*3,sizeof(float)*3,NY,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&y_cm_d, NY * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(y_cm_d,PITCH,y_cm[0],sizeof(float)*3,sizeof(float)*3,NY,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&x_triple_array_d, cnt_x * 3 * PITCH * sizeof(int))) printf("CUDA allocation error!\n");
    hipMemcpy2D(x_triple_array_d,PITCH,x_triple_array[0],sizeof(int)*3,sizeof(int)*3,cnt_x,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&y_triple_array_d, cnt_y * 3 * PITCH * sizeof(int))) printf("CUDA allocation error!\n");
    hipMemcpy2D(y_triple_array_d,PITCH,y_triple_array[0],sizeof(int)*3,sizeof(int)*3,cnt_y,hipMemcpyHostToDevice);  
  
    
    clock_gettime(CLOCK_REALTIME, &requestEnd);
    
    // call gpu and find triplets
    
    Triple * triple_array_d;   
    float * rmsd_array_d;
    
    size_t size = cnt_x * cnt_y * sizeof(Triple);
    size_t size_ratio = sizeof(Triple)/sizeof(float);

    
    if (size < MEM_SIZE_MAX) {
        if (hipSuccess != hipMalloc((void **)&triple_array_d, size)) printf("CUDA allocation error!\n");
        if (hipSuccess != hipMalloc((void**)&rmsd_array_d, cnt_x * cnt_y * sizeof(float))) printf("CUDA allocation error!\n");
    } else {
        if (hipSuccess != hipMalloc((void **)&triple_array_d, MEM_SIZE_MAX)) printf("CUDA allocation error!\n");
        if (hipSuccess != hipMalloc((void**)&rmsd_array_d, MEM_SIZE_MAX / size_ratio)) printf("CUDA allocation error!\n");
    }
 
         
    size_t free_mem, total_mem;
    size_t size_curr = size;
    size_t  y_triple_array_pos = 0;
    
    
    int not_enough_memory = 1;
    
    size_t cnt_y_rest = cnt_y;
    
    
    while (not_enough_memory) {
    
        clock_gettime(CLOCK_REALTIME, &requestStart);
        hipMemGetInfo(&free_mem, &total_mem);
        
        if (size_curr > MEM_SIZE_MAX){
        
            cnt_y = MEM_SIZE_MAX/(cnt_x * sizeof(Triple));
            cnt_y_rest -= cnt_y;
            
            size = MEM_SIZE_MAX;
            size_curr = cnt_x * cnt_y_rest * sizeof(Triple); 
        } else {
            cnt_y = cnt_y_rest;
            size = size_curr;
            not_enough_memory = 0;
        }
        
        int n_blocks_x = cnt_x/TILE_WIDTH + (cnt_x%TILE_WIDTH == 0 ? 0:1);
        int n_blocks_y = cnt_y/TILE_WIDTH + (cnt_y%TILE_WIDTH == 0 ? 0:1);
        dim3 numBlocks(n_blocks_x, n_blocks_y); 
        dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
        
        int * y_triple_array_d_curr = (int*)(((char*)y_triple_array_d) + (y_triple_array_pos * PITCH));
        
        find_triplets_gpu <<<numBlocks, threadsPerBlock>>> (x_d, x_cm_d, y_d, y_cm_d, NX, NY, x_triple_array_d, y_triple_array_d_curr, 
        cnt_x, cnt_y, triple_array_d, rmsd_array_d);
 
        hipDeviceSynchronize();
       
        
	Triple * triple_array_output_d;
    
        size_t no_of_out_pairs = TOP_RMSD < (cnt_x * cnt_y)? TOP_RMSD : cnt_x * cnt_y; 
        if (hipSuccess != hipMalloc((void **)&triple_array_output_d, no_of_out_pairs*sizeof(Triple))) printf("CUDA allocation error!\n");    
        
        thrust::device_vector<int>  indices(cnt_x * cnt_y); 
        thrust::sequence(indices.begin(),indices.end());
        thrust::device_ptr<Triple> structures(triple_array_d);
        thrust::device_ptr<Triple> structures_out(triple_array_output_d);
        thrust::device_ptr<float>rmsd(rmsd_array_d);
         
        //thrust::sort_by_key(rmsd, rmsd + cnt_x * cnt_y, structures);
        thrust::sort_by_key(rmsd, rmsd + cnt_x * cnt_y, indices.begin());
        

        thrust::device_vector<int>::iterator iter = indices.begin() + no_of_out_pairs;
        
        thrust::gather(indices.begin(), iter, structures, structures_out);

        // thrust::sort(structures, structures+ cnt_x * cnt_y, greater_rmsd());
        
        
        
        //hipMemcpy(triple_array, triple_array_d, no_of_out_pairs * sizeof(Triple), hipMemcpyDeviceToHost);
        hipMemcpy(triple_array, triple_array_output_d, no_of_out_pairs * sizeof(Triple), hipMemcpyDeviceToHost);
/*
        hipFree(triple_array_d);
        hipFree(y_triple_array_d);
        hipFree(rmsd_array_d);
*/
 
        
        int m;
        for(m = 0; m < no_of_out_pairs; ++m) {
            Insert(triple_array[m], *heap);
        }
    
 
        y_triple_array_pos += cnt_y;
        
        hipFree(triple_array_output_d);
        
    }
    
    
    hipFree(x_d);
    hipFree(y_d);
    hipFree(x_cm_d);
    hipFree(y_cm_d);
    hipFree(x_triple_array_d);
    hipFree(y_triple_array_d);
    
    hipFree(triple_array_d);

    hipFree(rmsd_array_d);
    
    free(triple_array);
    free_fmatrix(x);
    free_fmatrix(x_cm);
    free_fmatrix(y);
    free_fmatrix(y_cm);
   
    return 0;
} 

#ifdef	__cplusplus
}
#endif
