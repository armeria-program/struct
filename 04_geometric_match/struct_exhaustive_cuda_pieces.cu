#include "hip/hip_runtime.h"
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/system_error.h>



#ifdef	__cplusplus
extern "C" {
#endif

#include <hip/hip_runtime.h>   
#include "struct.h"

#define TILE_WIDTH 16
#define PITCH 64
#define MEM_SIZE_MAX 134217728L
#define CLOCK_PRECISION  1E9
/****************************************************************************************/
/****************************************************************************************/
/****************************************************************************************/
/****************************************************************************************/
/* opt_quat for gpu                             */

/**********************************************************************************/
__device__ int gauss_elim_gpu(float A[4][4],  float solution[4]) {

    float fMaxElem;
    float fAcc;
    float pfVect[4] = {0.0};
    int nDim = 4;
    int i, j, k, m;

    for(k=0; k<(nDim-1); k++){ // base row of matrix
  
	// search of line with max element
	fMaxElem = fabs( A[k][k] );
	m = k;
	for(i=k+1; i<nDim; i++) {
	    if(fMaxElem < fabs(A[i][k]) )  {
		fMaxElem = A[i][k];
		m = i;
	    }
	}
	// permutation of base line (index k) and max element line(index m)
	if(m != k) {
	    for(i=k; i<nDim; i++) {
		fAcc          = A[k][i];
		A[k][i] = A[m][i];
		A[m][i] = fAcc;
	    }
	    fAcc = pfVect[k];
	    pfVect[k] = pfVect[m];
	    pfVect[m] = fAcc;
	}

	if( A[k][k] == 0.) return 1; // needs improvement !!!

	// triangulation of matrix with coefficients
	for(j=(k+1); j<nDim; j++) { // current row of matrix
    
	    fAcc = - A[j][k] / A[k][k];
	    for(i=k; i<nDim; i++) {
		A[j][i] = A[j][i] + fAcc*A[k][i];
	    }
	    pfVect[j] = pfVect[j] + fAcc*pfVect[k]; // free member recalculation
	}
    }
    // in our case the last row should be all zeros
    solution[nDim-1] = 1.0; // arbitrary - we'll renormalize later
    float norm = 1.0;
    for (k=nDim-2; k>=0; k--)  {
	float sol = 0.0;
	for(i= k+1; i<nDim; i++){
	   sol  -= A[k][i]*solution[i];
	}
	sol = sol/A[k][k];
	solution[k] = sol;
	norm += sol*sol;
    }
    norm = sqrt(norm);
    for (k=0; k<nDim; k++) {
	solution[k] /= norm;
    }
    return 0;
}


/**********************************************/
__device__ int swap_gpu (float *a, float *b){
    float c = *a;
    *a = *b;
    *b =  c;
    return 0;
}

/**********************************************/
__device__ int  quadraticSolve_gpu(float *B, float *C,
			       float * root1, float * root2) {
    float discriminant = (*B * *B) - (4 * *C);
      
    //Cannot do imaginary numbers, yet
    if (discriminant < 0) return 1;
      
    float t = -0.5 * ( *B + ((*B < 0) ? -1 : 1) * sqrt(discriminant));
      
    *root1 = t;
    *root2 = *C / t;

    return 0;
}
/**********************************************/
__device__ int  cubicSolve_gpu(float p, float q, float r, 
			   float * root1, float * root2, float * root3) {
    
    float maxSqrt     = sqrt(FLT_MAX);
    
    if (r == 0) {
	//no ant term, so divide by x and the result is a
	//quadratic, but we must include the trivial x = 0 root
	if (quadraticSolve_gpu(&p,&q, root1, root2))
	{
	    *root3 = 0;
	    if (*root1 < *root2) swap_gpu(root1,root2);
	    if (*root2 < 0) 
	    {
		swap_gpu (root2,root3);
		if (*root1 < 0) swap_gpu(root1,root2);
	    }
	    return 3;
	}
	else
	{
	   *root1 = 0;
	    return 1;
	}
    }

    if ((p == 0) && (q == 0))
    {
	//Special case
	//Equation is x^3 == -r
	*root1 = *root2 = *root3 = pow(-r, (float)(1.0/3.0));
	return 3;
    }

    if ((p > maxSqrt) || (p < -maxSqrt))
    {
	//Equation limits to x^3 + p * x^2 == 0
	*root1 = -p;
	return 1;
    }

    if (q > maxSqrt)
    {
	//Special case, if q is large and the root is -r/q,
	//The x^3 term is negligble, and all other terms cancel.
	*root1 = -r / q;
	return 1;
    }

    if (q < -maxSqrt)
    {
	//Special case, equation is x^3 + q x == 0
	*root1 = -sqrt(-q);
	return 1;
    }

    if ((r > maxSqrt) || (r < -maxSqrt))
    {
	//Another special case
	//Equation is x^3 == -r
	*root1 = -pow(r, (float)(1.0/3.0));
	return 1;
    }

    float v = r + (2.0 * p * p / 9.0 - q) * (p / 3.0);

    if ((v > maxSqrt) || (v < -maxSqrt))
    {
	*root1 = -p;
	return 1;
    }

    float uo3 = q / 3.0 - p * p / 9.0;
    float u2o3 = uo3 + uo3;
      
    if ((u2o3 > maxSqrt) || (u2o3 < -maxSqrt))
    {
	if (p==0)
	{
	    if (q > 0)
	    {
		*root1 = -r / q;
		return 1;
	    }
	      
	    if (q < 0)
	    {
		*root1 = -sqrt(-q);
		return 1;
	    }
		
	    *root1 = 0;
	    return 1;
	}

	*root1 = -q/p;
	return 1;
    }

    float uo3sq4 = u2o3 * u2o3;
    if (uo3sq4 > maxSqrt)
    {
	if (p == 0)
	{
	    if (q > 0)
	    {
		*root1 = -r / q;
		return 1;
	    }

	    if (q < 0)
	    {
		*root1 = -sqrt(-q);
		return 1;
	    }

	    *root1 = 0;
	    return 1;
	}

	*root1 = -q / p;
	return 1;
    }

    float j = (uo3sq4 * uo3) + v * v;
  
    if (j > 0)  {//Only one root (but this test can be wrong due to a
	//catastrophic cancellation in j  (i.e. (uo3sq4 * uo3) == v * v)
	float w = sqrt(j);
	if (v < 0)
	    *root1 =         pow((float)(0.5*(w-v)), (float)(1.0/3.0))
 	           - (uo3) * pow((float)(2.0/(w-v)), (float)(1.0/3.0)) - p / 3.0;
	else
	    *root1 = uo3 * pow((float)(2.0/(w+v)), (float)(1.0/3.0))
	         	  -pow((float)(0.5*(w+v)), (float)(1.0/3.0)) - p/3.0;


	return 1;
    }
  
    if (uo3 >= 0) {//Multiple root detected	  
	*root1 = *root2 = *root3 = pow(v, (float)(1.0/3.0)) - p / 3.0;
	return 3;
    }

    float muo3 = - uo3;
    float s;
    if (muo3 > 0)
    {
	s = sqrt(muo3);
	if (p > 0) s = -s;
    }
    else
	s = 0;
      
    float scube = s * muo3;
    if (scube == 0)
    {
	*root1 = - p / 3.0;
	return 1;
    }
      
    float t = - v / (scube + scube);
    float k = acos(t) / 3.0;
    float cosk = cos(k);
    *root1 = (s + s) * cosk - p / 3.0;
      
    float sinsqk = 1.0 - cosk * cosk;
    if (sinsqk < 0) return 1;

    float rt3sink = sqrt(3.0) * sqrt(sinsqk);
    *root2 = s * (-cosk + rt3sink) - p / 3.0;
    *root3 = s * (-cosk - rt3sink) - p / 3.0;

 
    return 3;
}




/**********************************************/
__device__ int quadSolve_gpu(float C, float B, float A, float* root1, float* root2) {
    // Contingency: if A = 0, not a quadratic = linear
    if(A == 0) {
	//If B is zero then we have a NaN
	if(B == 0) return 0;
      
	*root1 = -1.0 * C / B;
	*root2 = *root1;
    }

    float discriminant = (B * B) - (4 * A * C);
      
    //Cannot do imaginary numbers, yet
    if (discriminant < 0) return 0;
      
    float t = -0.5 * ( B + ((B < 0) ? -1 : 1) * sqrt(discriminant));
      
    *root1 = t / A;
    *root2 = C / t;

    return 1;
}
/**************************************************/
__device__ float quarticError_gpu(float a, float b, float c, float d,
		    float roots[4], int  rootCount) {
    float errors[4];
    int root;
    float max_root = FLT_MIN;
    for ( root = 0; root < rootCount; ++ root) {
	
	float value = (((roots[root]+a) * roots[root] + b) * roots[root] + c) * roots[root] + d;

	if (value == 0) { errors[root] = 0; continue; }

	float deriv = ((4 * roots[root] + 3 * a) * roots[root] + 2 * b) * roots[root] + c;
      
	if (deriv != 0) { 
	    errors[root] = abs(value / deriv);
	} else {
	    float secDeriv = (12 * roots[root] + 6 * a) * roots[root] + 2 * b;
	    if (secDeriv != 0)
		errors[root] = sqrt(abs(value / secDeriv));
	    else
	    {
		float thirdDeriv = 24 * roots[root] + 6 * a;
		if (thirdDeriv != 0)
		    errors[root] = pow(abs(value / thirdDeriv), (float)(1.0/3.0));
		else
		    errors[root] = sqrt(sqrt(abs(value)/24));
	    }
	}
	if (max_root < errors[root]) max_root = errors[root];
    }

    return max_root;
}

/**********************************************/
__device__ int ferrariQuarticSolve_gpu(float a, float b, float c, float d,
			float * root1, float * root2, float * root3, float * root4) {
    float rts[4];
    float worst3[3];
    float qrts[4][3]; /* quartic roots for each cubic root */

    if (d == 0.0) {
	*root1 = 0.0;
	return cubicSolve_gpu(a,b,c,root2,root3,root4) + 1;
    }

    int   j;
    int   n4[4];
    float asqinv4;
    float ainv2;
    float d4;
    float yinv2;
    float v1[4],v2[4],v3[4];
    float p,q,r;
    float y;
    float e,f,esq,fsq,ef;
    float g,gg,h,hh;

    ainv2 = a*0.5;
    asqinv4 = ainv2*ainv2;
    d4 = d*4.0 ;

    p = b;
    q = a*c-d4;
    r = (asqinv4 - b)*d4 + c*c;
    int n3 = cubicSolve_gpu (p, q, r, &v3[0],&v3[1],&v3[2]);
    int j3;
    for (j3 = 0; j3 < n3; ++j3)
    {
	y = v3[j3];
	yinv2 = y*0.5;
	esq = asqinv4 - b - y;
	fsq = yinv2*yinv2 - d;
	if ((esq < 0.0) && (fsq < 0.0))
	    n4[j3] = 0;
	else
	{
	    ef = -(0.25*a*y + 0.5*c);
	    if ( ((a > 0.0)&&(y > 0.0)&&(c > 0.0))
		 || ((a > 0.0)&&(y < 0.0)&&(c < 0.0))
		 || ((a < 0.0)&&(y > 0.0)&&(c < 0.0))
		 || ((a < 0.0)&&(y < 0.0)&&(c > 0.0))
		 ||  (a == 0.0)||(y == 0.0)||(c == 0.0))
		/* use ef - */
	    {
		if ((b < 0.0)&&(y < 0.0))
		{
		    e = sqrt(esq);
		    f = ef/e;
		}
		else if (d < 0.0)
		{
		    f = sqrt(fsq);
		    e = ef/f;
		}
		else
		{
		    if (esq > 0.0)
			e = sqrt(esq);
		    else
			e = 0.0;
		    if (fsq > 0.0)
			f = sqrt(fsq);
		    else
			f = 0.0;
		    if (ef < 0.0)
			f = -f;
		}
	    }
	    else
		/* use esq and fsq - */
	    {
		if (esq > 0.0)
		    e = sqrt(esq);
		else
		    e = 0.0;
		if (fsq > 0.0)
		    f = sqrt(fsq);
		else
		    f = 0.0;
		if (ef < 0.0)
		    f = -f;
	    }
	    /* note that e >= 0.0 */
	    g = ainv2 - e;
	    gg = ainv2 + e;
	    if ( ((b > 0.0 && y > 0.0))
		 || ((b < 0.0 && y < 0.0)) )
	    {
		if ((a > 0.0  && e > 0.0)
		    || (a < 0.0  && e < 0.0) )
		    g = (b + y)/gg;
		else
		    if ((a > 0.0  && e < 0.0)
			|| (a < 0.0  && e > 0.0) )
			gg = (b + y)/g;
	    }
	    hh = -yinv2 + f;
	    h = -yinv2 - f;
	    if ( ((f > 0.0 && y < 0.0))
		 || ((f < 0.0 && y > 0.0)) )
		h = d/hh;
	    else
		if ( ((f < 0.0 && y < 0.0))
		     || ((f > 0.0 && y > 0.0)) )
		    hh = d/h;

	    int n1 = quadSolve_gpu(hh, gg, 1.0, &v1[0], &v1[1]);
	    int n2 = quadSolve_gpu(h,   g, 1.0, &v2[0], &v2[1]);
	    n4[j3] = n1*2+n2*2;
	    qrts[0][j3] = v1[0];
	    qrts[1][j3] = v1[1];
	    qrts[n1*2+0][j3] = v2[0];
	    qrts[n1*2+1][j3] = v2[1];
	}
	for (j = 0; j < n4[j3]; ++j)
	    rts[j] = qrts[j][j3];

	worst3[j3] = quarticError_gpu(a, b, c, d, rts, n4[j3]);
    } /* j3 loop */

    j3 = 0;
    if (n3 != 1)
    {
	if ((n4[1] > n4[j3]) ||
	    ((worst3[1] < worst3[j3] ) && (n4[1] == n4[j3]))) j3 = 1;

	if ((n4[2] > n4[j3]) ||
	    ((worst3[2] < worst3[j3] ) && (n4[2] == n4[j3]))) j3 = 2;
    }

    *root1 = qrts[0][j3];
    *root2 = qrts[1][j3];
    *root3 = qrts[2][j3];
    *root4 = qrts[3][j3];

    return (n4[j3]);
}

/*******************************************************************************/
__device__ int construct_ATA_gpu (float ATA[4][4], float a[3], float  b[3]){

    int i,j,k;
    float A[4][4] = {{ 0.0, -b[0], -b[1], -b[2]},
		      {b[0],   0.0, -a[2],  a[1]},
		      {b[1],  a[2],   0.0, -a[0]},
		      {b[2], -a[1],  a[0],   0.0}};
    
    for (i=0; i<4; i++ ) {
	for (j=0; j<4; j++ ) {
	    ATA[i][j] = 0.0;
	    for (k=0; k<4; k++ ) {
		ATA[i][j] += A[k][i]*A[k][j];
	    }
	}
    }
     
    return 0;
}

/*******************************************************************************/
__device__ int add_matrices_gpu  (float matrix1[4][4],float matrix2[4][4],
		   float result[4][4]){
    int i,j;

    for (i=0; i<4; i++ ) {
	for (j=0; j<4; j++ ) {
	    result[i][j] = matrix1[i][j] + matrix2[i][j];
	}
    }
    return 0;

}

/**********************************************************/
/**********************************************************/
__device__ int opt_quat_gpu(float *x_d, float *x_cm_d, float *y_d, float *y_cm_d,
			    int *set_of_directions_x, int *set_of_directions_y,
			    float q[4], float * rmsd) {

    
    float * x_sub[3];
    float * y_sub[3];
    int  ctr;
    int  i, j;
 
    float ATA     [4][4] = {{0.0}};
    float prev_ATA[4][4] = {{0.0}};
    float ATA_sum [4][4] = {{0.0}};
    float a[3] = {0.0}, b[3] = {0.0};
    
    //int add_matrices  (float matrix1[4][4],float matrix2[4][4],
    //		       float result[4][4]);
    //int construct_ATA (float ATA[4][4], float a[3], float  b[3]);


    if (!3) {
	*rmsd = -1;
	return 1;
    }
    memset ( &(q[0]), 0, 4*sizeof(float) );

    /* find the subset */
    for ( ctr =0; ctr < 3; ctr++ ) {
        i = set_of_directions_x[ctr];
        x_sub[ctr] =  (float*)(((char*)x_d) + (i * PITCH));
        i = set_of_directions_y[ctr];
        y_sub[ctr] =  (float*)(((char*)y_d) + (i * PITCH));
    }  

    /* B = ATA_sum matrix to diagonalize in order to get the quaternion */
    for ( ctr =0; ctr < 3; ctr++ ) {
   	for (i=0; i<3; i++ ) {
	    a[i] = y_sub[ctr][i] + x_sub[ctr][i];
	    b[i] = y_sub[ctr][i] - x_sub[ctr][i];
	}
 	construct_ATA_gpu (ATA, a, b);
	add_matrices_gpu (prev_ATA, ATA, ATA_sum);
	memcpy (prev_ATA[0], ATA_sum[0], 4*4*sizeof(float));
    }
    for (i=0; i<4; i++ ) {
	for (j=0; j<4; j++ ) {
	    ATA_sum[i][j] /= 3;
	}
    }

    /* diagonalize ATA_sum - the eigenvector corresponsing to the
       smallest lambda is the quaternion we are looking for; the
       eigenvalue is the rmsd*/
    float w[4];
    {
	float a,b,c,d,  e,f,g,h,  i,j,k,l,  m,n,o,p; /* synomyms*/
	a =  ATA_sum[0][0]; b =  ATA_sum[0][1]; c =  ATA_sum[0][2]; d =  ATA_sum[0][3]; 
	e =  ATA_sum[1][0]; f =  ATA_sum[1][1]; g =  ATA_sum[1][2]; h =  ATA_sum[1][3]; 
	i =  ATA_sum[2][0]; j =  ATA_sum[2][1]; k =  ATA_sum[2][2]; l =  ATA_sum[2][3]; 
	m =  ATA_sum[3][0]; n =  ATA_sum[3][1]; o =  ATA_sum[3][2]; p =  ATA_sum[3][3]; 

	float B, C, D, E; /* coefficients id the characteristic polynomial - according to sympy */
	/* A = 1.0; will not be used below */
	B = (-a - f - k - p);
	C =  a*f + a*k + a*p - b*e - c*i - d*m + f*k + f*p - g*j - h*n + k*p - l*o;
	D =  - a*f*k - a*f*p + a*g*j + a*h*n - a*k*p + a*l*o + b*e*k + b*e*p - b*g*i
	    - b*h*m - c*e*j + c*f*i + c*i*p - c*l*m - d*e*n + d*f*m - d*i*o + d*k*m
	    - f*k*p + f*l*o + g*j*p - g*l*n - h*j*o + h*k*n;
	E =   a*f*k*p - a*f*l*o - a*g*j*p + a*g*l*n + a*h*j*o - a*h*k*n - b*e*k*p
	    + b*e*l*o + b*g*i*p - b*g*l*m - b*h*i*o + b*h*k*m + c*e*j*p - c*e*l*n
	    - c*f*i*p + c*f*l*m + c*h*i*n - c*h*j*m - d*e*j*o + d*e*k*n + d*f*i*o
	    - d*f*k*m - d*g*i*n + d*g*j*m;

	ferrariQuarticSolve_gpu(B, C, D, E, &w[0], &w[1], &w[2], &w[3]);
    }

    
    float min_w = w[0];
    for (i=1; i<4; i++) {
	if (w[i] > min_w) continue;
	min_w = w[i];
    }
    
    for (i=0; i<4; i++) ATA_sum[i][i] -= min_w;
    if (gauss_elim_gpu (ATA_sum, q)) {
	return 1;
    }
    
    /* rmsd is the lowest eignevalue */
    *rmsd = sqrt(min_w);


    
    return 0;
}



/******************************************************************************/    
/******************************************************************************/    
/******************************************************************************/    
struct greater_rmsd{
    
    __host__ __device__
    bool operator()(Triple x, Triple y) 
    {
        return x.rmsd < y.rmsd;
    }
            
};
    
    
__device__ int  sum_to_zero_gpu (float *a, float *b ) {
    int i;
    float sum,aux;

    sum = 0;
    for (i=0; i<3; i++ ) {
	aux = a[i] + b[i];
	sum += aux*aux;
    }
    return (sum < 0.001);
}


__device__ int  normalized_cross_gpu (float *x, float *y, float * v, float *norm_ptr) {

    /* v is the output */
    float norm = 0;
    float vec[3];
    int i;

    if (sum_to_zero_gpu (x, y) ) return 1;
    
    vec[0] = x[1]*y[2] -  x[2]*y[1];
    norm  += vec[0]*vec[0];
    vec[1] = x[2]*y[0] -  x[0]*y[2]; 
    norm  += vec[1]*vec[1];
    vec[2] = x[0]*y[1] -  x[1]*y[0];
    norm  += vec[2]*vec[2];
    norm   = sqrt (norm);

    for (i=0; i<3; i++ ) {
	v[i] = vec[i] /norm;
    }

    if ( norm_ptr) *norm_ptr = norm;
    

    return 0;
    
}

__device__ int unnorm_dot_gpu (float *x, float *y, float * dot) {


    float cosine = 0;
    int i;
    
    for (i=0; i<3; i++ ) {
	cosine += x[i]*y[i];
    }

    if (cosine > 1.0 ) 
	cosine = 1.0; /* this should be numerical */
   
    *dot = cosine;
   
    return 0;
    
}


__device__ int distance_of_nearest_approach_gpu(float *x_d, float *x_cm_d, float *y_d,
						float *y_cm_d, int *set_of_directions_x,
						int *set_of_directions_y,
        int set_size, float * rmsd_ptr) {

    float cm_vector[3], distance_x, distance_y;
    float aux, rmsd;
    int i, ray_a, ray_b, a, b, prev_next, norm;
    float cross[3];

    //if (set_size <= 1) return 1;
       
    float * row_data_a, * row_data_b;

    
    rmsd = 0.0;
    norm = 0;
    /* the rmsd for the remaining vectors is ... */
    for (a = 0; a < set_size; a++) {

        for (prev_next = -1; prev_next <= +1; prev_next += 2) {
            b = (set_size + a + prev_next) % set_size;

            ray_a = set_of_directions_x[a];
            ray_b = set_of_directions_x[b];
            
            row_data_a = (float*)(((char*)x_cm_d) + (ray_a * PITCH));
            row_data_b = (float*)(((char*)x_cm_d) + (ray_b * PITCH));
    
            
            /* distance of nearest approach of ray b
               to the cm of a, in the set of directions x */
            
            for (i = 0; i < 3; ++i) {
                cm_vector[i] = row_data_b[i] - row_data_a[i];
            }
            
            
            row_data_a = (float*)(((char*)x_d) + (ray_a * PITCH));
            normalized_cross_gpu(cm_vector, row_data_a, cross, &distance_x);

            ray_a = set_of_directions_y[a];
            ray_b = set_of_directions_y[b];
            /* distance of nearest approach of ray b
               to the cm of a, in the set of directions y */
            
            row_data_a = (float*)(((char*)y_cm_d) + (ray_a * PITCH));
            row_data_b = (float*)(((char*)y_cm_d) + (ray_b * PITCH));
    
            for (i = 0; i < 3; ++i) {
                cm_vector[i] = row_data_b[i] - row_data_a[i];
            }
            
            row_data_a = (float*)(((char*)y_d) + (ray_a * PITCH));
            normalized_cross_gpu(cm_vector, row_data_a, cross, &distance_y);

            aux = distance_x - distance_y;
            rmsd += aux*aux;
            norm++;
        }

    }

    rmsd /= norm;
    rmsd = sqrt(rmsd);

    *rmsd_ptr = rmsd;

    return 0;
}
/***************************************************************************/
__device__ int  same_hand_triple_gpu(float *x_d, float *x_cm_d, float *y_d, float *y_cm_d,
				     int *set_of_directions_x, int *set_of_directions_y, int set_size){

    float cm_vector[3], avg_cm[3], cross[3], dx, dy;
    int i, ray_a, ray_b, ray_c;

    // if (set_size != 3) return 0;

    
    /*****************************/
    /*****************************/
    ray_a = set_of_directions_x[0];
    ray_b = set_of_directions_x[1];
    ray_c = set_of_directions_x[2];
    /* I better not use the cross prod here: a small diff
       int he angle makeing them pointing toward each other or
       away from each other changes the direction of cross prod;
       rather, use one vector, and distance between the cm's as the other */
    float * row_data_a, * row_data_b, * row_data_c;

    row_data_a = (float*)(((char*)x_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)x_d) + (ray_b * PITCH));
   
    
    normalized_cross_gpu(row_data_a, row_data_b, cross, NULL);
    
    row_data_a = (float*)(((char*)x_cm_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)x_cm_d) + (ray_b * PITCH));
    row_data_c = (float*)(((char*)x_cm_d) + (ray_c * PITCH)); 
    
    /* note I am making another cm vector here */
    for (i = 0; i < 3; i++) {
        avg_cm[i] = (row_data_b[i] - row_data_a[i])/2;
        cm_vector[i] = row_data_c[i] - avg_cm[i];
        //avg_cm[i] = (x_cm_d[ray_b][i] + x_cm_d[ray_a][i]) / 2;
        //cm_vector[i] = x_cm_d[ray_c][i] - avg_cm[i];
    }
    unnorm_dot_gpu(cm_vector, cross, &dx);

    /*****************************/
    /*****************************/
    ray_a = set_of_directions_y[0];
    ray_b = set_of_directions_y[1];
    ray_c = set_of_directions_y[2];
    
    row_data_a = (float*)(((char*)y_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)y_d) + (ray_b * PITCH));
   
    normalized_cross_gpu(row_data_a, row_data_b, cross, NULL);
    
    row_data_a = (float*)(((char*)y_cm_d) + (ray_a * PITCH));
    row_data_b = (float*)(((char*)y_cm_d) + (ray_b * PITCH));
    row_data_c = (float*)(((char*)y_cm_d) + (ray_c * PITCH));    
    
    /* note I am making another cm vector here */
    for (i = 0; i < 3; i++) {
        avg_cm[i]    = (row_data_b[i] - row_data_a[i])/2;
        cm_vector[i] = row_data_c[i]- avg_cm[i];
        
        //avg_cm[i] = (y_cm_d[ray_b][i] + y_cm_d[ray_a][i]) / 2;
        //cm_vector[i] = y_cm_d[ray_c][i] - avg_cm[i];
    }
    /*note: unnorm_dot thinks it is getting unit vectors,
      and evrything that is >1 will be "rounded" to 1
      (similarly for -1) - it doesn't do the normalization itself*/
    
    unnorm_dot_gpu(cm_vector, cross, &dy);

    if (dx * dy < 0) return 0;


    return 1; /* this isn't err value - the handedness is the same */
}

/***************************************************************************/

__global__ void find_triplets_gpu (float *x_d, float *x_cm_d, float *y_d, float *y_cm_d,
				   int NX, int NY, int *x_triple_array_d, int *y_triple_array_d, 
				   int cnt_x, int cnt_y, Triple *triple_array_d, float * rmsd_array_d){
    
    int row = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int col = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int i;
    int *row_data;
    float rmsd = -5; 
    float q[4];
    int triple_x[3], triple_y[3];
    // double q_init[4] = {0.0}; // no change

    
    if (row < cnt_x && col < cnt_y) {
	
        triple_array_d[row*cnt_y + col].rmsd = BAD_RMSD + 1;
        rmsd_array_d[row*cnt_y + col] = BAD_RMSD + 1;
        row_data = (int*)(((char*)x_triple_array_d) + (row * PITCH));
        
        for (i = 0; i < 3; ++i) {
            triple_x[i] =  row_data[i];
        }
        
        row_data = (int*)(((char*)y_triple_array_d) + (col * PITCH));
        
        for (i = 0; i < 3; ++i) {
            triple_y[i] =  row_data[i];
        }  
        
        for (i = 0; i <3; ++i) {
            triple_array_d[row*cnt_y + col].triple_x[i] = triple_x[i];
            triple_array_d[row*cnt_y + col].triple_y[i] = triple_y[i];
        }

        if (!same_hand_triple_gpu(x_d, x_cm_d, y_d, y_cm_d, triple_x, triple_y, 3) ) return;

        distance_of_nearest_approach_gpu(x_d, x_cm_d, y_d, y_cm_d, triple_x, triple_y, 3, &rmsd);
        if (rmsd > BAD_RMSD) {
            return; 
        }

	if (opt_quat_gpu (x_d, x_cm_d, y_d, y_cm_d, triple_x, triple_y, q, &rmsd)) return;
	
	// insert values to array of structs
        triple_array_d[row*cnt_y + col].rmsd = rmsd;
        rmsd_array_d[row*cnt_y + col] = rmsd;
        
    }
    
    
}

extern int insert_triple_to_heap_gpu(Representation* X_rep, Representation* Y_rep,
				     int ** x_triple_array, int ** y_triple_array, int x_triple_cnt,
				     int y_triple_cnt, PriorityQueue * heap) {
    

    double **x_db = X_rep->full;
    double **x_cm_db = X_rep->cm;
    double **y_db = Y_rep->full;
    double **y_cm_db = Y_rep->cm;
    int NX = X_rep->N_full;
    int NY = Y_rep->N_full;
    
    float **x = fmatrix(NX, 3);
    float **x_cm = fmatrix(NX, 3);;
    float **y = fmatrix(NY, 3);;
    float **y_cm = fmatrix(NY, 3);;
    
    int i,j;
    
    for(i = 0; i < NX; ++i) {
        for(j = 0; j < 3; ++j) {
            x[i][j] = x_db[i][j];
            x_cm[i][j] = x_cm_db[i][j];
        }
    }
    
    for(i = 0; i < NY; ++i) {
        for(j = 0; j < 3; ++j) {
            y[i][j] = y_db[i][j];
            y_cm[i][j] = y_cm_db[i][j];
        }
    }
    
    float *x_d;
    float *x_cm_d;
    float *y_d;
    float *y_cm_d;
    
    int *x_triple_array_d;
    int *y_triple_array_d;
    
    
    int cnt_x = x_triple_cnt;
    int cnt_y = y_triple_cnt;
    
    struct timespec requestStart, requestEnd;
    clock_gettime(CLOCK_REALTIME, &requestStart);
        
    
    Triple * triple_array = (Triple *) malloc(TOP_RMSD * sizeof(Triple));
    
    
    if (hipSuccess != hipMalloc(&x_d, NX * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(x_d,PITCH,x[0],sizeof(float)*3,sizeof(float)*3,NX,hipMemcpyHostToDevice);   
    
    if (hipSuccess != hipMalloc(&x_cm_d, NX * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(x_cm_d,PITCH,x_cm[0],sizeof(float)*3,sizeof(float)*3,NX,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&y_d, NY * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(y_d,PITCH,y[0],sizeof(float)*3,sizeof(float)*3,NY,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&y_cm_d, NY * 3 * PITCH * sizeof(float))) printf("CUDA allocation error!\n");
    hipMemcpy2D(y_cm_d,PITCH,y_cm[0],sizeof(float)*3,sizeof(float)*3,NY,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&x_triple_array_d, cnt_x * 3 * PITCH * sizeof(int))) printf("CUDA allocation error!\n");
    hipMemcpy2D(x_triple_array_d,PITCH,x_triple_array[0],sizeof(int)*3,sizeof(int)*3,cnt_x,hipMemcpyHostToDevice);
    
    if (hipSuccess != hipMalloc(&y_triple_array_d, cnt_y * 3 * PITCH * sizeof(int))) printf("CUDA allocation error!\n");
    hipMemcpy2D(y_triple_array_d,PITCH,y_triple_array[0],sizeof(int)*3,sizeof(int)*3,cnt_y,hipMemcpyHostToDevice);  
  
    
    clock_gettime(CLOCK_REALTIME, &requestEnd);
    
    // call gpu and find triplets
    
    Triple * triple_array_d;   
    float * rmsd_array_d;
    
    size_t size = cnt_x * cnt_y * sizeof(Triple);
    size_t size_ratio = sizeof(Triple)/sizeof(float);

    
    if (size < MEM_SIZE_MAX) {
        if (hipSuccess != hipMalloc((void **)&triple_array_d, size)) printf("CUDA allocation error!\n");
        if (hipSuccess != hipMalloc((void**)&rmsd_array_d, cnt_x * cnt_y * sizeof(float))) printf("CUDA allocation error!\n");
    } else {
        if (hipSuccess != hipMalloc((void **)&triple_array_d, MEM_SIZE_MAX)) printf("CUDA allocation error!\n");
        if (hipSuccess != hipMalloc((void**)&rmsd_array_d, MEM_SIZE_MAX / size_ratio)) printf("CUDA allocation error!\n");
    }
 
         
    size_t free_mem, total_mem;
    size_t size_curr = size;
    size_t  y_triple_array_pos = 0;
    
    
    int not_enough_memory = 1;
    
    size_t cnt_y_rest = cnt_y;
    
    
    while (not_enough_memory) {
    
        clock_gettime(CLOCK_REALTIME, &requestStart);
        hipMemGetInfo(&free_mem, &total_mem);
        
        if (size_curr > MEM_SIZE_MAX){
        
            cnt_y = MEM_SIZE_MAX/(cnt_x * sizeof(Triple));
            cnt_y_rest -= cnt_y;
            
            size = MEM_SIZE_MAX;
            size_curr = cnt_x * cnt_y_rest * sizeof(Triple); 
        } else {
            cnt_y = cnt_y_rest;
            size = size_curr;
            not_enough_memory = 0;
        }
        
        int n_blocks_x = cnt_x/TILE_WIDTH + (cnt_x%TILE_WIDTH == 0 ? 0:1);
        int n_blocks_y = cnt_y/TILE_WIDTH + (cnt_y%TILE_WIDTH == 0 ? 0:1);
        dim3 numBlocks(n_blocks_x, n_blocks_y); 
        dim3 threadsPerBlock(TILE_WIDTH, TILE_WIDTH);
        
        int * y_triple_array_d_curr = (int*)(((char*)y_triple_array_d) + (y_triple_array_pos * PITCH));
        
        find_triplets_gpu <<<numBlocks, threadsPerBlock>>> (x_d, x_cm_d, y_d, y_cm_d, NX, NY, x_triple_array_d, y_triple_array_d_curr, 
        cnt_x, cnt_y, triple_array_d, rmsd_array_d);
 
        hipDeviceSynchronize();
       
        
	Triple * triple_array_output_d;
    
        size_t no_of_out_pairs = TOP_RMSD < (cnt_x * cnt_y)? TOP_RMSD : cnt_x * cnt_y; 
        if (hipSuccess != hipMalloc((void **)&triple_array_output_d, no_of_out_pairs*sizeof(Triple))) printf("CUDA allocation error!\n");    
        
        thrust::device_vector<int>  indices(cnt_x * cnt_y); 
        thrust::sequence(indices.begin(),indices.end());
        thrust::device_ptr<Triple> structures(triple_array_d);
        thrust::device_ptr<Triple> structures_out(triple_array_output_d);
        thrust::device_ptr<float>rmsd(rmsd_array_d);
         
        //thrust::sort_by_key(rmsd, rmsd + cnt_x * cnt_y, structures);
        thrust::sort_by_key(rmsd, rmsd + cnt_x * cnt_y, indices.begin());
        

        thrust::device_vector<int>::iterator iter = indices.begin() + no_of_out_pairs;
        
        thrust::gather(indices.begin(), iter, structures, structures_out);

        // thrust::sort(structures, structures+ cnt_x * cnt_y, greater_rmsd());
        
        
        
        //hipMemcpy(triple_array, triple_array_d, no_of_out_pairs * sizeof(Triple), hipMemcpyDeviceToHost);
        hipMemcpy(triple_array, triple_array_output_d, no_of_out_pairs * sizeof(Triple), hipMemcpyDeviceToHost);
/*
        hipFree(triple_array_d);
        hipFree(y_triple_array_d);
        hipFree(rmsd_array_d);
*/
 
        
        int m;
        for(m = 0; m < no_of_out_pairs; ++m) {
            Insert(triple_array[m], *heap);
        }
    
 
        y_triple_array_pos += cnt_y;
        
        hipFree(triple_array_output_d);
        
    }
    
    
    hipFree(x_d);
    hipFree(y_d);
    hipFree(x_cm_d);
    hipFree(y_cm_d);
    hipFree(x_triple_array_d);
    hipFree(y_triple_array_d);
    
    hipFree(triple_array_d);

    hipFree(rmsd_array_d);
    
    free(triple_array);
    free_fmatrix(x);
    free_fmatrix(x_cm);
    free_fmatrix(y);
    free_fmatrix(y_cm);
   
    return 0;
}

    

    
#ifdef	__cplusplus
}
#endif
